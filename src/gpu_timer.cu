#include "gpu_timer.cuh"
#include "utils.cuh"

namespace GpuTimer
{
    void Timer::start()
    {
        CHECK_CUDA(hipEventCreate(&this->_start));
        CHECK_CUDA(hipEventCreate(&this->_end));
        CHECK_CUDA(hipEventRecord(this->_start));
    }

    void Timer::end()
    {
        CHECK_CUDA(hipEventRecord(this->_end));
        CHECK_CUDA(hipEventSynchronize(this->_end));
        CHECK_CUDA(hipEventElapsedTime(&this->_timeInMS, this->_start, this->_end));

        CHECK_CUDA(hipEventDestroy(this->_start));
        CHECK_CUDA(hipEventDestroy(this->_end));
    }

    void Timer::printResult(const char *s)
    {
        printf("[GPU TIMER] Step: \"%s\", Time: %f ms\n", s, this->_timeInMS);
    }
} // GpuTimer