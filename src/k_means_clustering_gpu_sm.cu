#include "hip/hip_runtime.h"
#include "k_means_clustering_gpu_sm.cuh"

namespace KMeansClusteringGPUSM
{
    __device__ float pointToClusterDistanceSquared(KMeansData::KMeansDataGPU d_data, size_t pointIndex, size_t clusterIndex)
    {
        float distance = 0;
        for (size_t d = 0; d < d_data.DIM; d++)
        {
            float diff = KMeansData::Helpers::GetCoord(d_data.d_pointsValues, d_data.pointsCount, pointIndex, d) - KMeansData::Helpers::GetCoord(d_data.d_clustersValues, d_data.clustersCount, clusterIndex, d);

            distance += diff * diff;
        }
        return distance;
    }

    __device__ size_t findNearestCluster(KMeansData::KMeansDataGPU d_data, size_t pointIndex)
    {
        float minDist = pointToClusterDistanceSquared(d_data, pointIndex, 0);
        size_t minDistIndex = 0;
        for (size_t j = 1; j < d_data.clustersCount; j++)
        {
            float distSquared = pointToClusterDistanceSquared(d_data, pointIndex, j);

            if (distSquared < minDist)
            {
                minDist = distSquared;
                minDistIndex = j;
            }
        }
        return minDistIndex;
    }

    // Function for finding new membership for each point
    // Each thread should be responsible for single point
    __global__ void calculateMembershipAndNewClusters(KMeansData::KMeansDataGPU d_data, float *d_newClusters, uint32_t *d_newClustersMembershipCount, size_t *d_memberships, int *d_shouldContinue)
    {
        auto threadId = blockDim.x * blockIdx.x + threadIdx.x;
        auto localThreadId = threadIdx.x;

        extern __shared__ int sharedMemory[];
        float *s_clusters = (float *)sharedMemory;
        uint32_t *s_clustersMembershipCount = (uint32_t *)&s_clusters[d_data.clustersCount * d_data.DIM];
        int *s_shouldContinue = (int *)&s_clustersMembershipCount[d_data.clustersCount];

        // Initialize shared memory in each block
        if (localThreadId == 0)
        {
            s_shouldContinue[0] = 0;
        }
        if (localThreadId < d_data.clustersCount * d_data.DIM)
        {
            s_clusters[localThreadId] = 0;
        }
        if (localThreadId < d_data.clustersCount)
        {
            s_clustersMembershipCount[localThreadId] = 0;
        }

        // Ensure shared memory is properly initialized
        __syncthreads();

        // For each point find its nearest cluster, update membership table and save results in shared memory
        if (threadId < d_data.pointsCount)
        {
            auto nearestClusterIndex = findNearestCluster(d_data, threadId);
            for (size_t d = 0; d < d_data.DIM; d++)
            {
                atomicAdd(&s_clusters[d * d_data.clustersCount + nearestClusterIndex], KMeansData::Helpers::GetCoord(d_data.d_pointsValues, d_data.pointsCount, threadId, d));
            }
            atomicAdd(&s_clustersMembershipCount[nearestClusterIndex], 1);
            auto previousClusterIndex = d_memberships[threadId];
            if (previousClusterIndex != nearestClusterIndex)
            {
                atomicAdd(&s_shouldContinue[0], 1);
                d_memberships[threadId] = nearestClusterIndex;
            }
        }

        // Finish all calculation made on shared memory
        __syncthreads();

        // Copy results from shared memory to global memory
        if (localThreadId == 0)
        {
            d_shouldContinue[blockIdx.x] = s_shouldContinue[0];
        }

        if (localThreadId < d_data.clustersCount * d_data.DIM)
        {
            d_newClusters[blockIdx.x * d_data.clustersCount * d_data.DIM + localThreadId] = s_clusters[localThreadId];
        }

        if (localThreadId < d_data.clustersCount)
        {
            d_newClustersMembershipCount[blockIdx.x * d_data.clustersCount + localThreadId] = s_clustersMembershipCount[localThreadId];
        }
    }

    // Function for accumulating clusters memberships count
    // There should be thread for every cluster
    // We know it always will be run in single block
    __global__ void accumulateNewClustersMemerships(KMeansData::KMeansDataGPU d_data, size_t *d_clustersMembershipCount, uint32_t *d_newClustersMembershipCount, size_t previousBlocksCount)
    {
        auto threadId = threadIdx.x;
        d_clustersMembershipCount[threadId] = 0;
        // For each cluster we calculate how many points belong to it accumulating results from all blocks
        for (size_t b = 0; b < previousBlocksCount; b++)
        {
            d_clustersMembershipCount[threadId] += d_newClustersMembershipCount[d_data.clustersCount * b + threadId];
        }
    }

    // Function for updating clusters based on new membership
    // There should be thread spawned for every cluster for every dimension, so CLUSTERS_COUNT * DIM total
    // We know it always will be run in single block
    __global__ void updateClusters(KMeansData::KMeansDataGPU d_data, size_t *d_clustersMembershipCount, float *d_newClusters, size_t previousBlocksCount)
    {
        auto threadId = threadIdx.x;
        d_data.d_clustersValues[threadId] = 0;
        // For each cluster dimension we accumulate results from all blocks
        for (size_t b = 0; b < previousBlocksCount; b++)
        {
            d_data.d_clustersValues[threadId] += d_newClusters[d_data.clustersCount * d_data.DIM * b + threadId];
        }
        size_t clusterId = threadId % d_data.clustersCount;
        // We divide by number of points in cluster to get mean
        d_data.d_clustersValues[threadId] /= d_clustersMembershipCount[clusterId];
    }

    Utils::ClusteringResult kMeansClustering(KMeansData::KMeansDataGPU d_data)
    {
        CpuTimer::Timer cpuTimer;
        GpuTimer::Timer gpuTimer;

        // PointsCount is always greater than dim * clustersCount * newClustersBlockCount (~ 20 * 20 * 1000 = 400 000 << 1 000 000 )
        const uint32_t newClustersBlocksCount = ceil(d_data.pointsCount * 1.0 / Consts::THREADS_PER_BLOCK);
        const size_t newClustersSharedMemorySize = d_data.clustersCount * d_data.DIM * sizeof(float) + d_data.clustersCount * sizeof(uint32_t) + sizeof(int);

        // We want to have clustersCount threads
        // We know in worse case scenario it's 20 threads < 1024
        const uint32_t accumulateNewClustersMemershipsBlocksCount = 1;

        // We want to have clustersCount * DIM threads
        // We know in worst case scenario it's 20 * 20 = 400 < 1024, so it's always gonna fit in one block
        const uint32_t updateClustersBlocksCount = 1;

        // Check if device has enough memory for our shared memory size
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        if (newClustersSharedMemorySize > prop.sharedMemPerBlock)
        {
            throw std::runtime_error("Required shared memory exceeds device limits");
        }

        size_t *d_memberships = nullptr;
        size_t *d_clustersMembershipCount = nullptr;
        float *d_newClusters = nullptr;
        uint32_t *d_newClustersMembershipCount = nullptr;
        int *d_shouldContinue = nullptr;
        int *shouldContinue = nullptr;

        // Prepare memory for storing results on CPU side
        thrust::host_vector<float> clustersValues(d_data.clustersCount * d_data.DIM);
        thrust::host_vector<size_t> membership(d_data.pointsCount);

        bool isError = false;
        std::runtime_error error("placeholder");

        try
        {
            // GPU allocations
            CHECK_CUDA(hipMalloc(&d_memberships, sizeof(size_t) * d_data.pointsCount));
            // We initialize the array that membership[i] = size_t::MAX
            CHECK_CUDA(hipMemset(d_memberships, 0xFF, sizeof(size_t) * d_data.pointsCount));

            CHECK_CUDA(hipMalloc(&d_clustersMembershipCount, sizeof(size_t) * d_data.clustersCount));

            // We have separate clustersValues for each block
            CHECK_CUDA(hipMalloc(&d_newClusters, sizeof(float) * d_data.clustersCount * d_data.DIM * newClustersBlocksCount));

            // We have separate clustersCount for each block
            CHECK_CUDA(hipMalloc(&d_newClustersMembershipCount, sizeof(uint32_t) * d_data.clustersCount * newClustersBlocksCount));

            CHECK_CUDA(hipMalloc(&d_shouldContinue, sizeof(int) * newClustersBlocksCount));

            // CPU allocation
            shouldContinue = (int *)malloc(sizeof(int) * newClustersBlocksCount);

            if (shouldContinue == nullptr)
            {
                throw std::runtime_error("Cannot allocate memory");
            }

            printf("[START] K-means clustering (main algorithm)\n");
            gpuTimer.start();
            // We don't need to call cudaDeviceSynchronzie because we use single device and we don't use cuda streams
            for (size_t k = 0; k < Consts::MAX_ITERATION; k++)
            {
                // Calculate new membership
                calculateMembershipAndNewClusters<<<newClustersBlocksCount, Consts::THREADS_PER_BLOCK, newClustersSharedMemorySize>>>(d_data, d_newClusters, d_newClustersMembershipCount, d_memberships, d_shouldContinue);
                CHECK_CUDA(hipGetLastError());
                CHECK_CUDA(hipDeviceSynchronize());

                // If all blocks return false than we know that no change was made and we can break from loop
                CHECK_CUDA(hipMemcpy(shouldContinue, d_shouldContinue, sizeof(int) * newClustersBlocksCount, hipMemcpyDeviceToHost));
                size_t totalShouldContinue = 0;
                for (size_t b = 0; b < newClustersBlocksCount; b++)
                {
                    totalShouldContinue += shouldContinue[b];
                }
                printf("[INFO] Iteration: %ld, changed points: %ld\n", k, totalShouldContinue);
                if (totalShouldContinue == 0)
                {
                    break;
                }

                // Accumulate counts from all blocks from previous kernel
                accumulateNewClustersMemerships<<<accumulateNewClustersMemershipsBlocksCount, d_data.clustersCount>>>(d_data, d_clustersMembershipCount, d_newClustersMembershipCount, newClustersBlocksCount);
                CHECK_CUDA(hipGetLastError());

                // Calculate new clusters
                updateClusters<<<updateClustersBlocksCount, d_data.clustersCount * d_data.DIM>>>(d_data, d_clustersMembershipCount, d_newClusters, newClustersBlocksCount);
                CHECK_CUDA(hipGetLastError());
            }
            gpuTimer.end();
            gpuTimer.printResult("K-means clustering (main algorithm)");

            // Wait for GPU to finish calculations
            CHECK_CUDA(hipDeviceSynchronize());

            // Copy result from GPU to CPU
            printf("[START] Copy data from GPU to CPU\n");
            cpuTimer.start();
            CHECK_CUDA(hipMemcpy(clustersValues.data(), d_data.d_clustersValues, sizeof(float) * clustersValues.size(), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(membership.data(), d_memberships, sizeof(size_t) * d_data.pointsCount, hipMemcpyDeviceToHost));
            cpuTimer.end();
            cpuTimer.printResult("Copy data from GPU to CPU");
        }
        catch (const std::runtime_error &e)
        {
            fprintf(stderr, "[ERROR]: %s", e.what());
            isError = true;
            error = e;
            goto ERROR_HANDLING;
        }

    ERROR_HANDLING:
        // GPU deallocations
        if (d_memberships != nullptr)
        {
            hipFree(d_memberships);
        }
        if (d_clustersMembershipCount != nullptr)
        {
            hipFree(d_clustersMembershipCount);
        }
        if (d_newClusters != nullptr)
        {
            hipFree(d_newClusters);
        }
        if (d_newClustersMembershipCount != nullptr)
        {
            hipFree(d_newClustersMembershipCount);
        }
        if (d_shouldContinue != nullptr)
        {
            hipFree(d_shouldContinue);
        }
        if (d_data.d_pointsValues != nullptr)
        {
            hipFree(d_data.d_pointsValues);
        }
        if (d_data.d_clustersValues != nullptr)
        {
            hipFree(d_data.d_clustersValues);
        }

        // CPU deallocation
        if (shouldContinue != nullptr)
        {
            free(shouldContinue);
        }

        if (isError)
        {
            throw error;
        }

        return Utils::ClusteringResult{
            .clustersValues = clustersValues,
            .membership = membership,
        };
    }

    Utils::ClusteringResult kMeansClusteringMPI(const KMeansData::KMeansData &h_kMeansData)
    {
        CpuTimer::Timer cpuTimer;
        GpuTimer::Timer gpuTimer;

        // Initialize MPI environment
        int rank, size;
        MPI_Init(NULL, NULL);
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &size);

        printf("[INFO] Process %d of %d started\n", rank, size);

        // Get data dimensions
        size_t pointsCount = h_kMeansData.getPointsCount();
        size_t clustersCount = h_kMeansData.getClustersCount();
        size_t DIM = h_kMeansData.getDIM();

        // Calculate how many points each process will handle
        size_t pointsPerProcess = pointsCount / size;
        size_t remainingPoints = pointsCount % size;

        // Calculate start and end indices for this process
        size_t startPointIdx = rank * pointsPerProcess + (rank < remainingPoints ? rank : remainingPoints);
        size_t localPointsCount = pointsPerProcess + (rank < remainingPoints ? 1 : 0);

        printf("[INFO] Process %d handles points %zu to %zu\n", rank, startPointIdx, startPointIdx + localPointsCount - 1);

        // Prepare to distribute data
        thrust::host_vector<float> localPointsValues(localPointsCount * DIM);

        // On root process, prepare send counts and displacements
        thrust::host_vector<int> sendcounts(size);
        thrust::host_vector<int> displs(size);
        if (rank == 0)
        {
            int displacement = 0;
            for (int i = 0; i < size; i++)
            {
                size_t pointsForProcess = pointsPerProcess + (i < remainingPoints ? 1 : 0);
                sendcounts[i] = pointsForProcess * DIM;
                displs[i] = displacement;
                displacement += pointsForProcess * DIM;
            }
        }

        // Distribute point data - only rank 0 provides the source data
        MPI_Scatterv(
            rank == 0 ? h_kMeansData.getValues().data() : nullptr,
            rank == 0 ? sendcounts.data() : nullptr,
            rank == 0 ? displs.data() : nullptr,
            MPI_FLOAT,
            localPointsValues.data(),
            localPointsCount * DIM,
            MPI_FLOAT,
            0, MPI_COMM_WORLD);

        // For cluster centers, broadcast to all processes
        thrust::host_vector<float> clustersValues;
        if (rank == 0)
        {
            // Use the initial cluster values from the host data
            clustersValues = h_kMeansData.getClustersValues();
        }
        else
        {
            // Other processes allocate memory to receive the data
            clustersValues.resize(clustersCount * DIM);
        }

        // Broadcast initial cluster centers to all processes
        MPI_Bcast(clustersValues.data(), clustersCount * DIM, MPI_FLOAT, 0, MPI_COMM_WORLD);

        // Set up GPU data
        KMeansData::KMeansDataGPU d_data;
        d_data.DIM = DIM;
        d_data.clustersCount = clustersCount;
        d_data.pointsCount = localPointsCount;

        // GPU allocations - now only allocate and transfer the local subset
        CHECK_CUDA(hipMalloc(&d_data.d_pointsValues, sizeof(float) * localPointsCount * DIM));
        CHECK_CUDA(hipMemcpy(d_data.d_pointsValues, localPointsValues.data(),
                              sizeof(float) * localPointsCount * DIM, hipMemcpyHostToDevice));

        CHECK_CUDA(hipMalloc(&d_data.d_clustersValues, sizeof(float) * clustersCount * DIM));
        CHECK_CUDA(hipMemcpy(d_data.d_clustersValues, clustersValues.data(),
                              sizeof(float) * clustersCount * DIM, hipMemcpyHostToDevice));

        // Other GPU memory allocations
        const uint32_t newClustersBlocksCount = ceil(localPointsCount * 1.0 / Consts::THREADS_PER_BLOCK);
        const size_t newClustersSharedMemorySize = clustersCount * DIM * sizeof(float) +
                                                   clustersCount * sizeof(uint32_t) + sizeof(int);

        // Resource allocation
        size_t *d_memberships = nullptr;
        size_t *d_clustersMembershipCount = nullptr;
        float *d_newClusters = nullptr;
        uint32_t *d_newClustersMembershipCount = nullptr;
        int *d_shouldContinue = nullptr;
        int *shouldContinue = nullptr;

        bool isError = false;
        std::runtime_error error("placeholder");

        try
        {
            // Allocate device memory
            CHECK_CUDA(hipMalloc(&d_memberships, sizeof(size_t) * localPointsCount));
            CHECK_CUDA(hipMemset(d_memberships, 0xFF, sizeof(size_t) * localPointsCount));

            CHECK_CUDA(hipMalloc(&d_clustersMembershipCount, sizeof(size_t) * clustersCount));
            CHECK_CUDA(hipMalloc(&d_newClusters, sizeof(float) * clustersCount * DIM * newClustersBlocksCount));
            CHECK_CUDA(hipMalloc(&d_newClustersMembershipCount, sizeof(uint32_t) * clustersCount * newClustersBlocksCount));
            CHECK_CUDA(hipMalloc(&d_shouldContinue, sizeof(int) * newClustersBlocksCount));

            // CPU buffers for communication
            shouldContinue = (int *)malloc(sizeof(int) * newClustersBlocksCount);
            thrust::host_vector<size_t> localMembership(localPointsCount);
            thrust::host_vector<float> localClusterSums(clustersCount * DIM, 0.0f);
            thrust::host_vector<size_t> localClusterCounts(clustersCount, 0);
            thrust::host_vector<float> globalClusterSums(clustersCount * DIM, 0.0f);
            thrust::host_vector<size_t> globalClusterCounts(clustersCount, 0);

            if (shouldContinue == nullptr)
            {
                throw std::runtime_error("Cannot allocate memory");
            }

            printf("[START] Process %d: K-means clustering (main algorithm)\n", rank);
            gpuTimer.start();

            bool continueIterating = true;
            int globalContinue = 1;

            for (size_t k = 0; k < Consts::MAX_ITERATION && continueIterating; k++)
            {
                // Calculate new membership
                calculateMembershipAndNewClusters<<<newClustersBlocksCount, Consts::THREADS_PER_BLOCK, newClustersSharedMemorySize>>>(
                    d_data, d_newClusters, d_newClustersMembershipCount, d_memberships, d_shouldContinue);
                CHECK_CUDA(hipGetLastError());
                CHECK_CUDA(hipDeviceSynchronize());

                // Check if points changed clusters
                CHECK_CUDA(hipMemcpy(shouldContinue, d_shouldContinue, sizeof(int) * newClustersBlocksCount, hipMemcpyDeviceToHost));
                int localShouldContinue = 0;
                for (size_t b = 0; b < newClustersBlocksCount; b++)
                {
                    localShouldContinue += shouldContinue[b];
                }

                // Gather local cluster information
                accumulateNewClustersMemerships<<<1, clustersCount>>>(
                    d_data, d_clustersMembershipCount, d_newClustersMembershipCount, newClustersBlocksCount);
                CHECK_CUDA(hipGetLastError());

                // Extract local cluster sums - gather raw sums before calculating new centers
                // We need raw sums for MPI reduction
                size_t *h_clustersMembershipCount = localClusterCounts.data();
                CHECK_CUDA(hipMemcpy(h_clustersMembershipCount, d_clustersMembershipCount,
                                      sizeof(size_t) * clustersCount, hipMemcpyDeviceToHost));

                // Get local cluster sums from GPU
                float *h_localClusterSums = localClusterSums.data();
                for (size_t c = 0; c < clustersCount; c++)
                {
                    for (size_t d = 0; d < DIM; d++)
                    {
                        float sum = 0.0f;
                        for (size_t b = 0; b < newClustersBlocksCount; b++)
                        {
                            CHECK_CUDA(hipMemcpy(&sum, &d_newClusters[b * clustersCount * DIM + d * clustersCount + c],
                                                  sizeof(float), hipMemcpyDeviceToHost));
                            h_localClusterSums[c * DIM + d] += sum;
                        }
                    }
                }

                // Use MPI to combine results across all processes
                MPI_Allreduce(&localShouldContinue, &globalContinue, 1, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
                MPI_Allreduce(localClusterSums.data(), globalClusterSums.data(), clustersCount * DIM,
                              MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);
                MPI_Allreduce(localClusterCounts.data(), globalClusterCounts.data(), clustersCount,
                              MPI_UNSIGNED_LONG, MPI_SUM, MPI_COMM_WORLD);

                // Calculate new cluster centers based on global data
                for (size_t c = 0; c < clustersCount; c++)
                {
                    if (globalClusterCounts[c] > 0)
                    {
                        for (size_t d = 0; d < DIM; d++)
                        {
                            clustersValues[c * DIM + d] = globalClusterSums[c * DIM + d] / globalClusterCounts[c];
                        }
                    }
                }

                // Update clusters on all GPUs with the new centroids
                CHECK_CUDA(hipMemcpy(d_data.d_clustersValues, clustersValues.data(),
                                      sizeof(float) * clustersCount * DIM, hipMemcpyHostToDevice));

                // Check if we should continue iterating
                continueIterating = (globalContinue > 0);

                // Reset local sums and counts for next iteration
                std::fill(localClusterSums.begin(), localClusterSums.end(), 0.0f);
                std::fill(localClusterCounts.begin(), localClusterCounts.end(), 0);

                printf("[INFO] Process %d: Iteration %ld, global changed points: %d\n", rank, k, globalContinue);
            }

            gpuTimer.end();
            if (rank == 0)
            {
                gpuTimer.printResult("K-means clustering (main algorithm)");
            }

            // Gather final memberships
            CHECK_CUDA(hipMemcpy(localMembership.data(), d_memberships,
                                  sizeof(size_t) * localPointsCount, hipMemcpyDeviceToHost));

            // Prepare to gather all memberships to rank 0
            thrust::host_vector<size_t> globalMembership;
            thrust::host_vector<int> recvCounts;
            thrust::host_vector<int> displacements;

            if (rank == 0)
            {
                globalMembership.resize(pointsCount);
                recvCounts.resize(size);
                displacements.resize(size);

                int displacement = 0;
                for (int i = 0; i < size; i++)
                {
                    int localSize = pointsPerProcess + (i < remainingPoints ? 1 : 0);
                    recvCounts[i] = localSize;
                    displacements[i] = displacement;
                    displacement += localSize;
                }
            }

            // Gather all memberships to rank 0
            MPI_Gatherv(localMembership.data(), localPointsCount, MPI_UNSIGNED_LONG,
                        rank == 0 ? globalMembership.data() : nullptr,
                        rank == 0 ? recvCounts.data() : nullptr,
                        rank == 0 ? displacements.data() : nullptr,
                        MPI_UNSIGNED_LONG, 0, MPI_COMM_WORLD);

            // Create result
            Utils::ClusteringResult result;
            if (rank == 0)
            {
                result.clustersValues = clustersValues;
                result.membership = globalMembership;
            }

            // Cleanup resources
            if (d_memberships)
                hipFree(d_memberships);
            if (d_clustersMembershipCount)
                hipFree(d_clustersMembershipCount);
            if (d_newClusters)
                hipFree(d_newClusters);
            if (d_newClustersMembershipCount)
                hipFree(d_newClustersMembershipCount);
            if (d_shouldContinue)
                hipFree(d_shouldContinue);
            if (d_data.d_pointsValues)
                hipFree(d_data.d_pointsValues);
            if (d_data.d_clustersValues)
                hipFree(d_data.d_clustersValues);

            // CPU cleanup
            if (shouldContinue)
                free(shouldContinue);

            // Finalize MPI before returning
            MPI_Finalize();

            return result;
        }
        catch (const std::runtime_error &e)
        {
            fprintf(stderr, "[ERROR] Process %d: %s\n", rank, e.what());
            isError = true;
            error = e;

            // Cleanup resources
            if (d_memberships)
                hipFree(d_memberships);
            if (d_clustersMembershipCount)
                hipFree(d_clustersMembershipCount);
            if (d_newClusters)
                hipFree(d_newClusters);
            if (d_newClustersMembershipCount)
                hipFree(d_newClustersMembershipCount);
            if (d_shouldContinue)
                hipFree(d_shouldContinue);
            if (d_data.d_pointsValues)
                hipFree(d_data.d_pointsValues);
            if (d_data.d_clustersValues)
                hipFree(d_data.d_clustersValues);

            // CPU cleanup
            if (shouldContinue)
                free(shouldContinue);

            // Finalize MPI
            MPI_Finalize();

            throw error;
        }
    }

} // KMeansClusteringGPUSM
