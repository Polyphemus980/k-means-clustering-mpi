#include <iostream>
#include <hip/hip_runtime.h>
#include <exception>
#include <cstring>

#include "utils.cuh"
#include "k_means_data.cuh"
#include "k_means_clustering_cpu.cuh"
#include "file_io.cuh"

// This function is an actual entry point
// We assume that at this point `inputFile` is changed in a way that
// Only N, DIM and K was read from it
template <size_t DIM>
void start(FILE *inputFile, size_t pointsCount, size_t clustersCount, Utils::ProgramArgs &programArgs)
{
    auto h_kMeansData = FileIO::LoadFromTextFile<DIM>(inputFile, pointsCount, clustersCount);
    auto result = KMeansClusteringCPU::kMeanClustering(h_kMeansData);
}

int main(int argc, char **argv)
{
    if (argc != 5)
    {
        std::cout << "Invalid arguments count\n";
        Utils::usage(argv[0]);
    }

    Utils::InputFileType inputFileType{};
    if (strcmp(argv[1], "txt") == 0)
    {
        inputFileType = Utils::InputFileType::TEXT;
    }
    else if (strcmp(argv[1], "bin") == 0)
    {
        inputFileType = Utils::InputFileType::BINARY;
    }
    else
    {
        std::cout << "Invalid file type\n";
        Utils::usage(argv[0]);
    }

    Utils::AlgorithmMode algorithmMode{};
    if (strcmp(argv[2], "cpu") == 0)
    {
        algorithmMode = Utils::AlgorithmMode::CPU;
    }
    else if (strcmp(argv[2], "gpu1") == 0)
    {
        algorithmMode = Utils::AlgorithmMode::GPU_FIRST;
    }
    else if (strcmp(argv[2], "gpu2") == 0)
    {
        algorithmMode = Utils::AlgorithmMode::GPU_SECOND;
    }
    else
    {
        std::cout << "IValid algorithm mode\n";
        Utils::usage(argv[0]);
    }

    Utils::ProgramArgs args{
        .algorithmMode = algorithmMode,
        .inputFileType = inputFileType,
        .inputFilePath = argv[3],
        .outputFilePath = argv[4]};

    FILE *inputFile = fopen(args.inputFilePath, "r");

    Utils::Parameters parameters{};

    if (args.inputFileType == Utils::InputFileType::TEXT)
    {
        parameters = FileIO::loadParamsFromTextFile(inputFile);
    }

    switch (parameters.dimensions)
    {
    case 1:
        start<1>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 2:
        start<2>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 3:
        start<3>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 4:
        start<4>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 5:
        start<5>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 6:
        start<6>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 7:
        start<7>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 8:
        start<8>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 9:
        start<9>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 10:
        start<10>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 11:
        start<11>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 12:
        start<12>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 13:
        start<13>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 14:
        start<14>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 15:
        start<15>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 16:
        start<16>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 17:
        start<17>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 18:
        start<18>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 19:
        start<19>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    case 20:
        start<20>(inputFile, parameters.pointsCount, parameters.clustersCount, args);
        break;
    default:
        throw std::runtime_error("Unsupported dimension");
        break;
    }

    return 0;
}
